
#include <hip/hip_runtime.h>
//note: please do not modify this file manually!
//      this file has been generated automatically by BOAST version 2.1.0
//      by: make boast_kernels

/*
!=====================================================================
!
!          S p e c f e m 3 D  G l o b e  V e r s i o n  8 . 0
!          --------------------------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/

#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif

#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif

__global__ void compute_iso_kernel(const float * epsilondev_xx, const float * epsilondev_yy, const float * epsilondev_xy, const float * epsilondev_xz, const float * epsilondev_yz, const float * epsilon_trace_over_3, const float * b_epsilondev_xx, const float * b_epsilondev_yy, const float * b_epsilondev_xy, const float * b_epsilondev_xz, const float * b_epsilondev_yz, const float * b_epsilon_trace_over_3, float * mu_kl, float * kappa_kl, const int NSPEC, const float deltat){
  int ispec;
  int ijk_ispec;

  ispec = blockIdx.x + (blockIdx.y) * (gridDim.x);

  if (ispec < NSPEC) {
    ijk_ispec = threadIdx.x + (NGLL3) * (ispec);

    mu_kl[ijk_ispec] = mu_kl[ijk_ispec] + (deltat) * ((epsilondev_xx[ijk_ispec]) * (b_epsilondev_xx[ijk_ispec]) + (epsilondev_yy[ijk_ispec]) * (b_epsilondev_yy[ijk_ispec]) + (epsilondev_xx[ijk_ispec] + epsilondev_yy[ijk_ispec]) * (b_epsilondev_xx[ijk_ispec] + b_epsilondev_yy[ijk_ispec]) + ((epsilondev_xy[ijk_ispec]) * (b_epsilondev_xy[ijk_ispec]) + (epsilondev_xz[ijk_ispec]) * (b_epsilondev_xz[ijk_ispec]) + (epsilondev_yz[ijk_ispec]) * (b_epsilondev_yz[ijk_ispec])) * (2));

    kappa_kl[ijk_ispec] = kappa_kl[ijk_ispec] + (deltat) * (((epsilon_trace_over_3[ijk_ispec]) * (b_epsilon_trace_over_3[ijk_ispec])) * (9));
  }
}
